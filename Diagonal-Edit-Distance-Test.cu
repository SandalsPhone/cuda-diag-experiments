//TODO: TEST OUT IF THE SLICE AND Z FOR THE DIAGONAL IMPLEMENTATION WORKS AS INTENDED
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <string.h>

using namespace std;

__global__ void insertValues(char *X, char *Y, int *arr, int slice, int z, int rowLength, int colLength){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    //row gets shifted down depending on the value of z
    int row = id + z;

    int col;
    //col starts with slice until its higher than column length
    if(slice < colLength){
        col = slice - id;
    }
    else{
        col = (colLength - 1) - id;
    }

    //the position in 1D array is calculated with (row x column length) + column
    int pos = (row * colLength) + col;
    //arr[(row * colLength) + col] = slice;
    //printf("pos: (%d,%u) \n %i  \n", row, col, arr[(row*colLength) + col]);
    //printf("%i  \n", arr[(row*rowLength) + col]);


    if (col == 0) {
		arr[pos] = row;
	}
	else if (row == 0) {
		arr[pos] = col;
	}
	else if (X[col-1] == Y[row-1]) {
		arr[pos] = arr[pos - rowLength - 1];
        //arr[pos] = 2;
	}
	else {
	//dp[row - 1][col] og 2nd min
		arr[pos] = 1 + min(min(arr[pos - 1], arr[pos - rowLength]), arr[pos - rowLength - 1]);
        //arr[pos] = 1;
	}  
    
}

__global__ void printArr(int *arr, int rowLength, int colLength){
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    printf("Array from kernel:\n");
    for(int i= 0; i<rowLength; i++){
        for(int j= 0; j<colLength; j++){
            int pos = (i*colLength)+j;
            printf("%d:%i  ", pos, arr[(i*colLength) + j]);
        }
        printf("\n");
    }
    printf("\n");
}


int main(int argc, char *argv[]){
    //testing 2D arrays with 1D array representation too
    int *arr, *hostArr;


    char X[] = "ABCD";
    char Y[] = "CBAD";
    int rowLength = strlen(X);
    int colLength = strlen(Y);

    printf("X: %s\n", X);
    printf("Y: %s\n", Y);
    printf("X length: %i\n", rowLength);
    printf("Y length: %i\n", colLength);

    char *deviceX, *deviceY;
    hipMalloc(&deviceX, rowLength*sizeof(char));
    hipMalloc(&deviceY, colLength*sizeof(char));

    hipMemcpy(deviceX, X, rowLength*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(deviceY, Y, colLength*sizeof(char), hipMemcpyHostToDevice);

    rowLength++;
    colLength++;

    

    int size = rowLength*colLength;

    //allocate memory
    hostArr = (int*) std::malloc(size*sizeof(int));
    hipMalloc(&arr, size*sizeof(int));


    //print before running the kernel(s)
    printf("Array before:\n");
    for(int i= 0; i<rowLength; i++){
        for(int j= 0; j<colLength; j++){
            printf("%i  ", hostArr[(i*colLength) + j]);
        }
        printf("\n");
    }
    printf("\n");

    //just as a note:
    //the for loop below for the diagonal implementation uses slice as the baseline
    //slice functions as to determine the position of the diagonal for the iteration
    //example:
    //lets say slice = 1, visually on a 3x3 array its like this:
    //| 0  s  0 |
    //| s  0  0 |  <-- with s being the representation of the slice
    //| 0  0  0 |
    //
    //this continues until slice is higher the length of the column,
    //where the diagonal cannot be calculated with only the column as reference
    //the diagonal needs to continue through the 'bottom' half of the array 
    //
    //this is where z starts to function
    //after the slice is higher than the column, z is calculated using this formula:
    //z = slice - column + 1
    //lets say the slice is only 1 higher than the column,
    //this means the z is 1, and with that, the diagonal "shifts down" by 1
    //an example on a 3x3 array with z = 1:
    //| 0  0  0 |
    //| 0  0  s |
    //| 0  s  0 |

    int z, tSize, lowestLength;
    int bSize = 1;
    if(colLength<rowLength){
        lowestLength = colLength;
    }
    else{
        lowestLength = rowLength;
    }

	for(int slice=0; slice < colLength*2; slice++){
		if(slice < colLength){
			z = 0;
            if(slice<lowestLength){
                tSize = slice + 1;
            }
            else{
                tSize = lowestLength;
            }
		}
		else{
			z = slice - colLength + 1;
            tSize = colLength - z;
		}
		
        //calculate thread and blocks used
		if(tSize <= 256){
			bSize = 1;
		}
		else{
			bSize = tSize / 256;
            tSize = 256;
		}

		insertValues<<<bSize, tSize>>>(deviceX, deviceY, arr, slice, z, rowLength, colLength);
	}
    printf("\n");

    //copy device array from insertValues to host array
    hipMemcpy(hostArr, arr, size*sizeof(int), hipMemcpyDeviceToHost);

    //printArr<<<1,1>>>(arr, rowLength, colLength);
    
    //print after running the kernel(s)
    printf("Array after:\n");
    for(int i= 0; i<rowLength; i++){
        for(int j= 0; j<colLength; j++){
            //printf("%d,%u :%i  ", i, j, hostArr[(i*colLength) + j]);
            printf("%i  ", hostArr[(i*colLength) + j]);
        }
        printf("\n");
    }
    printf("\n");

    //free memory
    hipFree(arr);
    free(hostArr);

    return 0;
}
